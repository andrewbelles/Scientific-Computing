#include "hip/hip_runtime.h"
// 2D simulation
#include <GL/glew.h>

// Cuda headers
#include <GL/glext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <GL/glew.h>
#include <cuda_gl_interop.h>

// C++ headers
#include <hip/driver_types.h>
#include <iostream> 
#include <sstream>
#include <fstream>
#include <random>

// External libraries 
#include <stdexcept>
#include <thrust/device_ptr.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/sort.h>

// Graphics libraries 
#include <raylib.h>

// CPU Constants
constexpr size_t BLOCKSIZE      = 256;  
constexpr float L_host          = 5.0; 
constexpr float rho0_host       = 1.0;
constexpr float2 zero_vec_host{0.0, 0.0};
constexpr size_t MN_host        = 125;
constexpr float c0_host         = 7.0; 
constexpr float visc_host       = 1e-1;

// Matching GPU Constants 
__constant__ size_t MN          = 100;
__constant__ float L            = 5.0;
__constant__ float rho0         = 1.0; 
__constant__ float c0           = 7.0;
__constant__ float visc         = 1e-1;
__constant__ float2 zero_vector{0.0, 0.0};

extern "C" {
__constant__ float poly_C; 
__constant__ float spiky_C;
__constant__ float cubic_C; 
__constant__ float pres_floor;
}

__device__ __host__ float2 add_float2(float2 a, float2 b)
{
  return make_float2(a.x + b.x, a.y + b.y);
}

__device__ __host__ float2 subtract_float2(float2 a, float2 b)
{
  return make_float2(a.x - b.x, a.y - b.y);
}

struct ParticleMatrix 
{
  std::size_t cols; 
  float mass, h, *density;
  float2 *x, *v, *a; 
  float2 *fpres, *fvisc, *fsys;
};


// Macro for all Cuda API Calls to return error and function name etc. of offender 
#define CUDA_CHECK(call)                              \
    do {                                              \
        hipError_t err__ = (call);                   \
        if (err__ != hipSuccess) {                   \
            std::ostringstream ss;                    \
            ss << __FILE__ << ':' << __LINE__ << "  " \
               << hipGetErrorName(err__) << " – "    \
               << hipGetErrorString(err__);          \
            throw std::runtime_error(ss.str());       \
        }                                             \
    } while (0)


// Spatial lookup structure 
struct Spatial 
{
  struct Value 
  {
    uint2 cell_id; 
    uint64_t key;
    size_t pidx; 
  }; 

  // col number of entries 
  uint2 cells;
  Value *entries;
  size_t *start, *end;
};


// All metadata from device ptrs needed by CPU 
struct Metadata
{
  uint2 cells;
  size_t N; 
  float h;
};


// Compute absolute position in grid space and return to kernel
__device__ uint2 position_to_cell_id(float2 position, float smoothing_radius)
{
  uint2 cell_id;
  cell_id.x = std::floor(position.x / smoothing_radius);
  cell_id.y = std::floor(position.y / smoothing_radius);
  
  float position_y_rel = position.y / smoothing_radius; 

  if (position_y_rel < 0.0)
    printf("Negative Cell ID: (%f, %f) (%u,%u)\n", position.x, position.y, cell_id.x, cell_id.y);

  return cell_id; 
}


// Return a packed key from a key value pair. 
__host__ __device__ uint64_t pack_key(uint2 cell_id, size_t xcells) 
{
  return static_cast<uint64_t>(cell_id.x) * xcells + static_cast<uint64_t>(cell_id.y);
}


// Kernel to compute all particles cell_id 
__global__ void set_keys(ParticleMatrix* particles, const Metadata meta, Spatial::Value* entries, size_t* start, size_t* end)
{
  const size_t idx = threadIdx.x + blockDim.x * blockIdx.x; 
  if (idx >= particles->cols)
    return; 

  // Get position in register as const 
  const float2 position = particles->x[idx]; 
  const uint2 cell_id   = position_to_cell_id(position, particles->h);

  // Assign values to table 
  entries[idx].cell_id = cell_id;  // TODO: Needed? 
  entries[idx].key     = pack_key(cell_id, meta.cells.x);
  entries[idx].pidx    = idx; 
}


// Get the start and end arrays from key values generated 
__global__ void define_cell_ranges(Spatial::Value* entries, size_t* start, size_t* end, size_t N, size_t xcell)
{
  size_t idx = threadIdx.x + blockDim.x * blockIdx.x; 
  if (idx >= N)
    return; 

  const uint64_t key = entries[idx].key;

  // Set idx of start and end cell if non-matching to cell previous to it
  if (idx == 0 || key != entries[idx - 1].key)
    start[key] = idx;

  if (idx + 1 == N || key != entries[idx + 1].key)
    end[key] = idx + 1;
}


// CPU program to generate a spatial table given the current state of particles 
__host__ void generate_spatial_table(ParticleMatrix* particles, Spatial table, const Metadata meta)
{
  const size_t N = meta.N; 
  size_t C = meta.cells.x * meta.cells.y;
  hipMemset(table.start, 0, C * sizeof(size_t));
  hipMemset(table.end,   0, C * sizeof(size_t));
  // Kernel call to set cell_id 
  const size_t GRIDSIZE = (N + BLOCKSIZE - 1) / BLOCKSIZE; 
  set_keys<<<GRIDSIZE, BLOCKSIZE>>>(particles, meta, table.entries, table.start, table.end); 
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  // Sort entries
  thrust::device_ptr<Spatial::Value> table_entries(table.entries);
  // Lambda comparator for entries with priority to x coordinate then y, then pidx 
  thrust::sort(table_entries, table_entries + N, 
    [] __device__ (const Spatial::Value& a, const Spatial::Value& b)
    {
      return a.key < b.key;
    }
  );

  // Kernel call to set start and end cells 
  define_cell_ranges<<<GRIDSIZE, BLOCKSIZE>>>(table.entries, table.start, table.end, N, meta.cells.x); 
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
}


// Cell-block tiling algorithm to generate neighbor list
__global__ void neighbor_search(
    ParticleMatrix* particles, 
    const Metadata meta, 
    Spatial::Value *entries, 
    size_t* start,
    size_t* end, 
    int* neighbor_counts,
    int* neighbor_list)
{
  const size_t cx   = blockIdx.x;
  const size_t cy   = blockIdx.y;
  const size_t cidx = cy * gridDim.x + cx; 

  // Size as defined by grid size 
  extern __shared__ size_t shared_pidx[];
  // Start and end indices 
  const size_t start_index  = start[cidx];     // TODO: Are we certain this is in bounds of start/end 
  const size_t end_index    = end[cidx];

  // Local count of particles 
  const size_t tidx   = threadIdx.x; 
  size_t lcount  = (end_index > start_index) ? (end_index - start_index) : 0;

  // Exit block for invalid lcount 
  __syncthreads();

  if (lcount == 0)
    return; 

  for (size_t base = 0; base < lcount; base += blockDim.x)
  {
    size_t lpid = base + tidx;
    if (lpid < lcount)
      shared_pidx[tidx] = entries[start_index + lpid].pidx;
    __syncthreads();

    if (lpid < lcount) 
    {
        
      // Get position of center particle at idx 
      size_t idx = shared_pidx[tidx];
      float2 x   = particles->x[idx];  

      for (int dy = -1; dy <= 1; dy++)
      {
        // Relative y coordinate, check if in bounds 
        int yrel = cy + dy;
        if (yrel < 0 || yrel >= meta.cells.y)
          continue;

        for (int dx = -1; dx <= 1; dx++)
        {
          // Relative x coordinate
          int xrel = cx + dx;
          if (xrel < 0 || xrel >= meta.cells.x)
            continue;

          // Get relative cell from relative x and y coordinates 
          int rel_cell = xrel + yrel * meta.cells.x;
          size_t rel_start = start[rel_cell];
          size_t rel_end   = end[rel_cell];

          for (size_t p = rel_start; p < rel_end; p++)
          {
            int jdx = entries[p].pidx; 
            // Skip self
            //if (jdx == idx)
            //  continue;

            // relative position 
            float2 xj = particles->x[jdx];
            float2 d = make_float2(x.x - xj.x, x.y - xj.y);
            // Check within smoothing radius
            if (d.x * d.x + d.y * d.y > particles->h * particles->h)
              continue; 

            // Pointer is one more than current neighbor count 
            int ptr = atomicAdd(&neighbor_counts[idx], 1);
            // Check if less than max neighbor count and set 
            if (ptr < MN)
              neighbor_list[idx * MN + ptr] = jdx;
          }
        }
      }
    }
    __syncthreads();
  }
}


// Builds a neighbor list
__host__ void neighbor_host(ParticleMatrix* particles, Spatial table, const Metadata meta, int* neighbor_counts, int* neighbor_list)
{
  const size_t N = meta.N; 
  hipMemsetAsync(neighbor_counts, 0, N * sizeof(int), 0);    // Set to 0 agnostic to whether it's already been done
  
  // Call w/ 2D grid 
  dim3 GRID(meta.cells.x, meta.cells.y);

  // Ensure shared memory spreads well for each block 
  constexpr size_t RED_BLOCKSIZE = BLOCKSIZE / 2;
  const size_t shared_memory = RED_BLOCKSIZE * sizeof(size_t);
  neighbor_search<<<GRID, RED_BLOCKSIZE, shared_memory>>>(
    particles, 
    meta, 
    table.entries,
    table.start,
    table.end,
    neighbor_counts,
    neighbor_list
  );
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
}

// Smoothing Kernel functions 

// poly6 2d density kernel
// Since r isn't explicitly required, opted to only use the squared value of r 
__device__ float poly6(float sqr, float sqh)
{
  float sqd = sqh - sqr;
  // Outside influence - shouldn't occur
  if (sqd <= 0.0)
    return 0.0;

  return poly_C * sqd * sqd * sqd;
}


// Gradient of spiky kernel
__device__ float2 spiky_gradient(float2 r, float h) {
    float rlen = sqrtf(r.x*r.x + r.y*r.y);
    if (rlen == 0.0 || rlen > h) 
      return zero_vector;

    float t = (h - rlen);
    float coeff = -spiky_C * t*t / rlen;  
    return make_float2(r.x * coeff,
                       r.y * coeff);
}


// Laplacian of cubic spline smoothing kernel
__device__ float cubic_spline_laplacian(float2 r, float h)
{
    float rlen = sqrtf(r.x*r.x + r.y*r.y);
    if (rlen > h) 
      return 0.0;

    // 2D constant: 40/(π h^5)
    return cubic_C * (h - rlen);
}


// a tiny kernel, called once per frame:
__global__ void enforce_boundaries(ParticleMatrix* particles) {
  size_t idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx >= particles->cols) return;

  float2 x = particles->x[idx];
  float2 v = particles->v[idx];
  const float restitution = 0.98;

  // left/right
  if (x.x < 1e-3) 
  { 
    x.x =  1e-3;
    v.x =  std::abs(v.x) * restitution; 
  }
  else if (x.x > L - (1e-3))
  { 
    x.x = L - 1e-3;
    v.x = -std::abs(v.x) * restitution; 
  }
  
  // bottom/top
  if (x.y < 1e-3) 
  {
    x.y =  1e-3;
    v.y =  std::abs(v.y) * restitution; 
  }
  else if (x.y > L - (1e-3))
  { 
    x.y = L - (1e-3);
    v.y = -std::abs(v.y) * restitution; 
  }

  particles->x[idx] = x;
  particles->v[idx] = v;
}


// Computes all accumulated density for each particle 
__global__ void compute_densities(ParticleMatrix* particles, const int* neighbor_counts, const int* neighbor_list)
{
  const size_t idx = threadIdx.x + blockDim.x * blockIdx.x; 
  if (idx >= particles->cols)
    return;

  if (neighbor_counts[idx] == 0)
    printf("Particle %lu has an empty neighbor count\n", idx);

  float sum = 1e-4; 
  float sqh = particles->h * particles->h;
  // Iterate over num of neighbors 
  int cap = std::min(neighbor_counts[idx], static_cast<int>(MN));
  for (int num = 0; num < cap; num++)
  {
    // Idx of relative particle from list  
    int jdx   = neighbor_list[idx * MN + num];

    // Get distance 
    float2 d  = make_float2(
      particles->x[idx].x - particles->x[jdx].x,
      particles->x[idx].y - particles->x[jdx].y
    );

    // Compute density from kernel 
    float sqr = d.x * d.x + d.y * d.y; 
    sum += particles->mass * poly6(sqr, sqh);
  }

  // Set accumulated density 
  particles->density[idx] = sum;
}


// Simple function to return non-linear clamped pressure
__device__ float compute_pressure(float rho_i)
{
  const float B = rho0 * c0 * c0 / 7.0;
  float r = rho_i / rho0;
  return (r > 1.0) ? B * ((r*r*r*r*r*r*r) - 1.0) : pres_floor;
}


// Density precomputed - Compute all forces  
__global__ void compute_forces(ParticleMatrix* particles, const int* neighbor_counts, const int* neighbor_list)
{
  const size_t idx = threadIdx.x + blockDim.x * blockIdx.x; 
  if (idx >= particles->cols)
    return; 

  // Get local values for particle 
  const float2 x      = particles->x[idx];
  const float xrho    = particles->density[idx]; 

  float xpres = compute_pressure(xrho); 

  const float2 v      = particles->v[idx];

  particles->fsys[idx].y += particles->mass * -98100; 
  
  int cap = std::min(neighbor_counts[idx], static_cast<int>(MN));
  for (int num = 0; num < cap; num++)
  {
    // Get relative values for neighbor 
    size_t jdx  = neighbor_list[idx * MN + num];
    float2 xrel = particles->x[jdx];
    float2 dj   = make_float2(x.x - xrel.x, x.y - xrel.y);

    // Compute pressure from density for relative particle
    float jrho = particles->density[jdx];
    float jpres = compute_pressure(jrho);

    // Compute the pressure force 
    float a = -particles->mass * ((xpres / (xrho * xrho)) + (jpres / (jrho * jrho)));
    float2 gW = spiky_gradient(dj, particles->h);
    float2 av = make_float2(gW.x * a, gW.y * a);
    particles->fpres[idx] = add_float2(particles->fpres[idx], av);

    // Compute the viscosity force 
    float laplacian = cubic_spline_laplacian(dj, particles->h);
    float b = visc * particles->mass / jrho * laplacian;
    float2 relv = subtract_float2(particles->v[jdx], v);
    particles->fvisc[idx] = add_float2(particles->fvisc[idx], make_float2(relv.x * b, relv.y * b));
  }

  // Compute acceleration directly
  float2 ftotal = add_float2(particles->fpres[idx], add_float2(particles->fvisc[idx], particles->fsys[idx]));
  particles->a[idx] = make_float2(ftotal.x / xrho, ftotal.y / xrho);
}


// Computes first half of integration with half a timestep 
__global__ void verlet_kick(ParticleMatrix* particles, float half_dt)
{
  const size_t idx = threadIdx.x + blockDim.x * blockIdx.x; 
  if (idx >= particles->cols)
    return;

  particles->v[idx].x += particles->a[idx].x * half_dt;
  particles->v[idx].y += particles->a[idx].y * half_dt; 
}


// Computes second half after forces have been calculated 
__global__ void verlet_drift(ParticleMatrix* particles, float half_dt)
{
  const size_t idx = threadIdx.x + blockDim.x * blockIdx.x; 
  if (idx >= particles->cols)
    return;

  particles->x[idx].x += particles->v[idx].x * half_dt;
  particles->x[idx].y += particles->v[idx].y * half_dt;
}


__global__ void reset_accumulators(ParticleMatrix* particles)
{
  const size_t idx = threadIdx.x + blockDim.x * blockIdx.x; 
  if (idx >= particles->cols)
    return;

  //if (particles->x[idx].x > L || particles->x[idx].x < 0 || particles->x[idx].y > L || particles->x[idx].y < 0)
  //  printf("Particle out of bounds (%f,%f): %lu\n", particles->x[idx].x, particles->x[idx].y, idx);

  // Zero out accumulated values 
  particles->density[idx] = 1e-4;
  particles->fsys[idx] = particles->fpres[idx] = particles->fvisc[idx] = zero_vector; 
}


__host__ float adaptive_dt(ParticleMatrix* particles, const Metadata& meta)
{
  const float CFL    = 0.2; 
  const float CFORCE = 0.25;
  const float CVISC  = 0.125;

  // Reduce the max magnitude from across all particles efficiently 
  float max_velocity = thrust::transform_reduce(
    thrust::device,
    thrust::make_counting_iterator(0),
    thrust::make_counting_iterator(static_cast<int>(meta.N)), 
    [=] __device__ (int i) -> float
    {
      float2 v = particles->v[i];
      return std::sqrt(v.x * v.x + v.y * v.y);
    },
    0.0, 
    thrust::maximum<float>()
  );

  float max_accel = thrust::transform_reduce(
    thrust::device,
    thrust::make_counting_iterator(0),
    thrust::make_counting_iterator(static_cast<int>(meta.N)), 
    [=] __device__ (int i) -> float
    {
      float2 a = particles->a[i];
      return std::sqrt(a.x * a.x + a.y * a.y);
    },
    0.0, 
    thrust::maximum<float>()
  );

  float dt_acoustic = CFL * (meta.h / (c0_host + max_velocity));
  float dt_force    = CFORCE * std::sqrt(meta.h / (max_accel + 1e-3));
  float dt_visc     = CVISC * (meta.h * meta.h / (visc_host + 1e-3));
  
  // return smallest time constant
  return std::min(dt_acoustic, std::min(dt_force, dt_visc)); 
}

// Compute the forces given the filled neighbor counts and list
// Don't need table. 
__host__ void handle_forces(ParticleMatrix* particles, const Metadata& meta, int* neighbor_counts, int* neighbor_list)
{
  // TODO: Compute dynamic timestep 
  const float dt = adaptive_dt(particles, meta);
  const size_t GRIDSIZE = (meta.N + BLOCKSIZE - 1) / BLOCKSIZE;

  // KERNEL CALLS - All use the same launch parameters  
  reset_accumulators<<<GRIDSIZE, BLOCKSIZE>>>(particles);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  compute_densities<<<GRIDSIZE, BLOCKSIZE>>>(particles, neighbor_counts, neighbor_list);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  compute_forces<<<GRIDSIZE, BLOCKSIZE>>>(particles, neighbor_counts, neighbor_list);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  verlet_kick<<<GRIDSIZE, BLOCKSIZE>>>(particles, dt/2.0); 
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  
  verlet_drift<<<GRIDSIZE, BLOCKSIZE>>>(particles, dt);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  enforce_boundaries<<<GRIDSIZE, BLOCKSIZE>>>(particles);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  compute_densities<<<GRIDSIZE, BLOCKSIZE>>>(particles, neighbor_counts, neighbor_list);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  compute_forces<<<GRIDSIZE, BLOCKSIZE>>>(particles, neighbor_counts, neighbor_list);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  verlet_kick<<<GRIDSIZE, BLOCKSIZE>>>(particles, dt/2.0); 
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
}


namespace bufr {

// Struct to hold buffer 
struct Buffer 
{
  GLuint pos_vbo{0};
  GLuint rho_vbo{0};
  hipGraphicsResource* pos_res{nullptr};
  hipGraphicsResource* rho_res{nullptr};
};


// Takes number of particles and sets up buffers
void initialize_cuda_buffers(Buffer* buffers, size_t N)
{
  // Setup buffer for position and allocate resources from cuda 
  glGenBuffers(1, &buffers->pos_vbo);
  glBindBuffer(GL_ARRAY_BUFFER, buffers->pos_vbo);
  glBufferData(GL_ARRAY_BUFFER, N * sizeof(float2), nullptr, GL_DYNAMIC_DRAW); 
  glBindBuffer(GL_ARRAY_BUFFER, 0);
  hipGraphicsGLRegisterBuffer(&buffers->pos_res, buffers->pos_vbo, hipGraphicsRegisterFlagsNone);

  // And density 
  glGenBuffers(1, &buffers->rho_vbo);
  glBindBuffer(GL_ARRAY_BUFFER, buffers->rho_vbo);
  glBufferData(GL_ARRAY_BUFFER, N * sizeof(float), nullptr, GL_DYNAMIC_DRAW); 
  glBindBuffer(GL_ARRAY_BUFFER, 0);
  hipGraphicsGLRegisterBuffer(&buffers->rho_res, buffers->rho_vbo, hipGraphicsRegisterFlagsNone);
}


void update_buffers(Buffer* buffers, ParticleMatrix host_particles)
{
  size_t size; 
  float2* d_pos;
  float* d_rho;

  hipGraphicsMapResources(1, &buffers->pos_res, 0);
  hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&d_pos), &size, buffers->pos_res);
  hipMemcpy(d_pos, host_particles.x, host_particles.cols * sizeof(float2), hipMemcpyDeviceToDevice);
  hipGraphicsUnmapResources(1, &buffers->pos_res, 0);

  hipGraphicsMapResources(1, &buffers->rho_res, 0);
  hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&d_rho), &size, buffers->rho_res);
  hipMemcpy(d_rho, host_particles.density, host_particles.cols * sizeof(float), hipMemcpyDeviceToDevice);
  hipGraphicsUnmapResources(1, &buffers->rho_res, 0);
}


// Free allocated resourcess 
void unregister_buffers(Buffer* buffers)
{
  // Check for nullptr and unregister 
  if (buffers->pos_res != nullptr)
  {
    hipGraphicsUnregisterResource(buffers->pos_res);
    glDeleteBuffers(1, &buffers->pos_vbo);
    buffers->pos_res = nullptr;
  }

  // Again for density 
  if (buffers->rho_res != nullptr)
  {
    hipGraphicsUnregisterResource(buffers->rho_res);
    glDeleteBuffers(1, &buffers->rho_vbo);
    buffers->rho_res = nullptr;
  }
}

}

namespace shdr {

// Read all data from shader file into return value 
// Credit ChatGPT o4-mini-high
std::string load_shader(const std::string& path)
{
  std::ifstream shader_file(path, std::ios::in | std::ios::binary);
  if (!shader_file)
    throw std::runtime_error("File doesn't exist");

  std::string shader; 
  shader_file.seekg(0, std::ios::end);
  shader.resize(shader_file.tellg());
  shader_file.seekg(0, std::ios::beg);
  shader_file.read(&shader[0], shader.size());
  shader_file.close();

  return shader;
}


// Compile the shader
GLuint compile_shader(GLenum type, const std::string& src)
{
  char error[512];
  GLint status;
  GLuint shader = glCreateShader(type);
  const char* cstr = src.c_str();

  glShaderSource(shader, 1, &cstr, nullptr);
  glCompileShader(shader);
  glGetShaderiv(shader, GL_COMPILE_STATUS, &status);
  if (!status)
  {
    glGetShaderInfoLog(shader, 512, nullptr, error); 
    throw std::runtime_error(error);
  }

  return shader;
}


// Creates the program from file paths to fragment and vertex shaders 
GLuint create_program(const std::string& fragment_path, const std::string& vertex_path)
{
  // Get shaders from source and compile 
  std::string vertex_src   = load_shader(vertex_path);
  std::string fragment_src = load_shader(fragment_path);
  GLuint vertex_shader   = compile_shader(GL_VERTEX_SHADER, vertex_src);
  GLuint fragment_shader = compile_shader(GL_FRAGMENT_SHADER, fragment_src);
  GLuint program = glCreateProgram();

  GLint status;
  char error[512];

  // Attach shaders to program 
  glAttachShader(program, vertex_shader);
  glAttachShader(program, fragment_shader);
  glLinkProgram(program);

  glGetProgramiv(program, GL_LINK_STATUS, &status);
  if (!status)
  {
    glGetProgramInfoLog(program, 512, nullptr, error);
    glDeleteProgram(program);
    throw std::runtime_error(error);
  }

  glDeleteShader(vertex_shader);
  glDeleteShader(fragment_shader);

  return program;
}

}

// Main 
int main(void)
{
  // Create opengl context 
  InitWindow(800, 800, "SPH");
  glewInit();
  glEnable(GL_PROGRAM_POINT_SIZE);

  GLuint program = shdr::create_program("../shaders/fragment.frag", "../shaders/vertex.vert");
  GLint projection_location = glGetUniformLocation(program, "uProj");

  GLuint point_vao;
  glGenVertexArrays(1, &point_vao);
  glBindVertexArray(point_vao);

  // System constants
  constexpr size_t N = 8000; 

  constexpr float region_size = 2.5;
  constexpr float offset      = (L_host - region_size) / 2.0;

  constexpr float area = L_host * L_host / 2.0;
  constexpr float mass = rho0_host * area;
  constexpr float mass_per = mass / static_cast<float>(N);

  //const size_t Mx = static_cast<size_t>(std::ceil(std::sqrt(2.0 * static_cast<float>(N))));
  const size_t M    = static_cast<size_t>(std::ceil(std::sqrt(static_cast<float>(N))));
  const float delta = region_size / static_cast<float>(M);
  //const size_t My = static_cast<size_t>(std::ceil((L_host / 2.0) / delta));

  const float h = 1.3 * delta; 
  const float r = h * 0.8 * (600 / L_host);

  // Set metadata
  Metadata meta = (Metadata)
  {
    .cells = make_uint2(std::ceil(L_host / h), std::ceil(L_host / h)),
    .N     = N,
    .h     = h
  };

  // Allocate memory to particle device ptr
  ParticleMatrix h_p, *particles;
  hipMalloc(&h_p.density, N * sizeof(float));
  hipMalloc(&h_p.x, N * sizeof(float2)); 
  hipMalloc(&h_p.v, N * sizeof(float2));
  hipMalloc(&h_p.a, N * sizeof(float2));
  hipMalloc(&h_p.fpres, N * sizeof(float2)); 
  hipMalloc(&h_p.fvisc, N * sizeof(float2));
  hipMalloc(&h_p.fsys, N * sizeof(float2));
  hipMalloc(&particles, sizeof(ParticleMatrix));


  // Set constant values 
  h_p.cols = N;
  h_p.mass = mass_per; 
  h_p.h    = h;
  
  // Particle position initialization
  
  std::mt19937 rng(31);
  std::uniform_real_distribution<float> drift(-0.1f*delta, 0.1f*delta);

  std::vector<float2> host_positions;
  std::vector<float2> host_velocities(N, zero_vec_host);
  std::vector<float2> host_accelerations(N, zero_vec_host);

  host_positions.reserve(N);

  int pid = 0;
  for (size_t i = 0; i < M && pid < N; ++i) 
  {
    for (size_t j = 0; j < M && pid < N; ++j) 
    {
      float x = offset + (i + 0.5) * delta + drift(rng);
      float y = offset + (j + 0.5) * delta + drift(rng);

      // Clamp positions
      if (x < offset) 
        x = offset + 1e-4;
      else if (x > offset + region_size) 
        x = offset + region_size - 1e-4;

      if (y < offset) 
        y = offset + 1e-4;
      else if (y > offset + region_size) 
        y = offset + region_size - 1e-4;
      
      host_positions.emplace_back(make_float2(x, y));
      pid++;
    }
  }

  // Copy initialized vectors to device  
  hipMemcpy(h_p.x, host_positions.data(), N * sizeof(float2), hipMemcpyHostToDevice);
  hipMemcpy(h_p.v, host_velocities.data(), N * sizeof(float2), hipMemcpyHostToDevice);
  hipMemcpy(h_p.a, host_accelerations.data(), N * sizeof(float2), hipMemcpyHostToDevice);

  // Copy all to device pointer 
  hipMemcpy(particles, &h_p, sizeof(ParticleMatrix), hipMemcpyHostToDevice);

  // Table is already a host pointer so no memcpy required to a device pointer
  Spatial table;
  table.cells = meta.cells; 
  hipMalloc(&table.entries, N * sizeof(Spatial::Value));
  hipMalloc(&table.start, meta.cells.x * meta.cells.y * sizeof(size_t));
  hipMalloc(&table.end, meta.cells.x * meta.cells.y * sizeof(size_t));

  // Allocate memory for lists
  int *neighbor_list, *neighbor_counts; 
  hipMalloc(&neighbor_counts, N * sizeof(int));
  hipMalloc(&neighbor_list, N * MN_host * sizeof(int));

  // setting up matrix 
  float projection_matrix[16] = {
      2.0/L_host, 0,        0,  0,
      0,          2.0/L_host, 0,  0,
      0,          0,       -1,  0,
     -1.0,       -1.0,     0,   1
  };

  const float h8 = std::pow(h, 8);
  const float h5 = std::pow(h, 5);

  float POLY_CONST  = 4.0  / (M_PI * h8);
  float SPIKY_CONST = 30.0 / (M_PI * h5);
  float CUBIC_CONST = 40.0 / (M_PI * h5);
  constexpr float pres_floor_host = 0.01 * rho0_host * c0_host * c0_host;

  CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(poly_C), &POLY_CONST, sizeof(float)));
  CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(spiky_C), &SPIKY_CONST, sizeof(float)));
  CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cubic_C), &CUBIC_CONST, sizeof(float)));
  CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(pres_floor), &pres_floor_host, sizeof(float)));

  // Set up buffers 
  bufr::Buffer buffer;
  bufr::initialize_cuda_buffers(&buffer, N);

  // Simulation loop
  SetTargetFPS(144);
  while (!WindowShouldClose())
  {

    // One timestep of simulation  
    generate_spatial_table(particles, table, meta);
    neighbor_host(particles, table, meta, neighbor_counts, neighbor_list);
    handle_forces(particles, meta, neighbor_counts, neighbor_list);

    bufr::update_buffers(&buffer, h_p);

    // Handle drawing from buffer 

    BeginDrawing();

      ClearBackground(BLACK);

      // View port should match rectangle sizing 
      DrawRectangleLines(100, 100, 600, 600, WHITE);
      glViewport(100, 100, 600, 600);

      // Get position buffer 
      glBindVertexArray(point_vao);
      glUseProgram(program);
      glUniform1f(glGetUniformLocation(program, "pointSize"), r);
      // Map projection matrix for particle positions 
      glUniformMatrix4fv(projection_location, 1, GL_FALSE, projection_matrix);

      // Call buffers and draw points 
      glEnableVertexAttribArray(0);
      glBindBuffer(GL_ARRAY_BUFFER, buffer.pos_vbo);
      glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, 0, (void*)0);
      glDrawArrays(GL_POINTS, 0, N);

      // Disable buffers 
      glDisableVertexAttribArray(0);
      glBindBuffer(GL_ARRAY_BUFFER, 0);
      glBindVertexArray(0);
      glUseProgram(0);
      glViewport(0, 0, 800, 800);

    EndDrawing();
  }

  // Free resources 

  glDeleteProgram(program);
  CloseWindow();
  bufr::unregister_buffers(&buffer);

  hipFree(neighbor_counts);
  hipFree(neighbor_list); 
  hipFree(table.entries);
  hipFree(table.start);
  hipFree(table.end);

  // Copy particles back to host to free resources 
  hipMemcpy(&h_p, particles, sizeof(ParticleMatrix), hipMemcpyDeviceToHost);

  hipFree(h_p.x);
  hipFree(h_p.v);
  hipFree(h_p.a);
  hipFree(h_p.density);
  hipFree(h_p.fsys);
  hipFree(h_p.fpres);
  hipFree(h_p.fvisc);
  hipFree(particles);

  return 0; 
}
