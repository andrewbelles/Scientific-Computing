#include "hip/hip_runtime.h"

// 2D simulation
#include <GL/glew.h>

// Cuda headers
#include <GL/glext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <GL/glew.h>
#include <cuda_gl_interop.h>


// C++ headers
#include <iostream> 
#include <sstream>
#include <fstream>

// External libraries 
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

// Graphics libraries 
#include <raylib.h>

// Standard blocksize for kernel call 
constexpr size_t BLOCKSIZE = 256;  
constexpr float L_host = 1.0; 
constexpr float rho0_host = 1.0;
constexpr float2 zero_vec_host{0.0, 0.0};
constexpr size_t MN_host = 200;


// Max neighbor count
__constant__ size_t MN = 200;

__constant__ float L    = 1.0;
__constant__ float rho0 = 1.0; // kg/m^3 
__constant__ float c0   = 20.0;
__constant__ float visc = 1.0;    // TODO: What value? 

__constant__ float2 zero_vector{0.0, 0.0};

__device__ __host__ float2 add_float2(float2 a, float2 b)
{
  return make_float2(a.x + b.x, a.y + b.y);
}

__device__ __host__ float2 subtract_float2(float2 a, float2 b)
{
  return make_float2(a.x - b.x, a.y - b.y);
}

struct ParticleMatrix 
{
  std::size_t cols; 
  float mass, h, *density;
  float2 *x, *v; 
  float2 *fpres, *fvisc, *fsys;
};


// Macro for all Cuda API Calls to return error and function name etc. of offender 
#define CUDA_CHECK(call)                                                \
    do {                                                                \
        hipError_t err__ = (call);                                     \
        if (err__ != hipSuccess) {                                     \
            std::ostringstream ss;                                      \
            ss << __FILE__ << ':' << __LINE__ << "  "                   \
               << hipGetErrorName(err__) << " – "                      \
               << hipGetErrorString(err__);                            \
            throw std::runtime_error(ss.str());                         \
        }                                                               \
    } while (0)


// Spatial lookup structure 
struct Spatial 
{
  struct value 
  {
    uint2 cell_id; 
    uint64_t key;
    size_t pidx; 
  }; 

  // col number of entries 
  uint2 cells;
  value *entries;
  size_t *start, *end;

  // Set all values to size_t max in table for start and end arrays  
  void table_clear(size_t cell_count) {
    // each array is numCells long
    hipMemsetAsync(start, 0xFF, cell_count * sizeof(size_t), 0);
    hipMemsetAsync(end,   0xFF, cell_count * sizeof(size_t), 0);
  }

};


// All metadata from device ptrs needed by CPU 
struct Metadata
{
  uint2 cells;
  size_t N; 
};


// Compute absolute position in grid space and return to kernel
__device__ uint2 position_to_cell_id(float2 position, float smoothing_radius)
{
  uint2 cell_id;
  cell_id.x = std::floor(position.x / smoothing_radius);
  cell_id.y = std::floor(position.y / smoothing_radius);

  return cell_id; 
}


// Return a packed key from a key value pair. 
__host__ __device__ uint64_t pack_key(uint2 cell_id)
{
  // Morton style packing
  return (uint64_t(cell_id.x) << 32) | uint64_t(cell_id.y);
}


// Kernel to compute all particles cell_id 
__global__ void set_keys(ParticleMatrix* particles, Spatial::value* entries, size_t* start, size_t* end)
{
  const size_t idx = threadIdx.x + blockDim.x * blockIdx.x; 
  if (idx >= particles->cols)
    return; 

  // Get position in register as const 
  const float2 position = particles->x[idx]; 
  const uint2 cell_id   = position_to_cell_id(position, particles->h);

  // Assign values to table 
  entries[idx].cell_id = cell_id;  // TODO: Needed? 
  entries[idx].key     = pack_key(cell_id);
  entries[idx].pidx    = idx; 
}


// Get the start and end arrays from key values generated 
__global__ void define_cell_ranges(Spatial::value* entries, size_t* start, size_t* end, size_t N, size_t xcell)
{
  size_t idx = threadIdx.x + blockDim.x * blockIdx.x; 
  if (idx >= N)
    return; 

  const uint2 cell_id = entries[idx].cell_id; 
  const uint64_t key  = cell_id.y * xcell + cell_id.x; 

  // Set idx of start and end cell if non-matching to cell previous to it
  if (idx == 0 || key != entries[idx - 1].key)
    start[key] = idx;

  if (idx + 1 == N || key != entries[idx + 1].key)
    end[key] = idx + 1;
}


// CPU program to generate a spatial table given the current state of particles 
__host__ void generate_spatial_table(ParticleMatrix* particles, Spatial table, const Metadata meta)
{
  const size_t N = meta.N; 
  // if table is non-empty clear all values async 
  table.table_clear(meta.cells.x * meta.cells.y);

  // Kernel call to set cell_id 
  const size_t GRIDSIZE = (N + BLOCKSIZE - 1) / BLOCKSIZE; 
  set_keys<<<GRIDSIZE, BLOCKSIZE>>>(particles, table.entries, table.start, table.end); 
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  // Sort entries
  thrust::device_ptr<Spatial::value> table_entries(table.entries);
  // Lambda comparator for entries with priority to x coordinate then y, then pidx 
  thrust::sort(table_entries, table_entries + N, 
    [] __device__ (const Spatial::value& a, const Spatial::value& b)
    {
      return a.key < b.key;
    }
  );

  // Kernel call to set start and end cells 
  define_cell_ranges<<<GRIDSIZE, BLOCKSIZE>>>(table.entries, table.start, table.end, N, meta.cells.x); 
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
}


// Cell-block tiling algorithm to generate neighbor list
__global__ void neighbor_search(
    ParticleMatrix* particles, 
    const Metadata meta, 
    Spatial::value *entries, 
    size_t* start,
    size_t* end, 
    int* neighbor_counts,
    int* neighbor_list)
{
  const size_t bidx = blockIdx.x; 
  const size_t cx   = bidx % meta.cells.x;
  const size_t cy   = bidx / meta.cells.x;

  // Size as defined by grid size 
  extern __shared__ size_t shared_pidx[];
  // Start and end indices 
  const size_t start_index  = start[bidx];     // TODO: Are we certain this is in bounds of start/end 
  const size_t end_index    = end[bidx];

  // Local count of particles 
  const size_t tidx   = threadIdx.x; 
  size_t lcount  = (end_index > start_index) ? (end_index - start_index) : 0;

  // Exit block for invalid lcount 
  __syncthreads();

  if (lcount == 0)
    return; 

  for (size_t base = 0; base < lcount; base += blockDim.x)
  {
    size_t lpid = base + tidx;
    if (lpid < lcount)
      shared_pidx[tidx] = entries[start_index + lpid].pidx;
    __syncthreads();

    if (lpid < lcount) 
    {
        
      // Get position of center particle at idx 
      size_t idx = shared_pidx[tidx];
      float2 x   = particles->x[idx];  

      for (int dy = -1; dy <= 1; dy++)
      {
        // Relative y coordinate, check if in bounds 
        int yrel = cy + dy;
        if (yrel < 0 || yrel >= meta.cells.y)
          continue;

        for (int dx = -1; dx <= 1; dx++)
        {
          // Relative x coordinate
          int xrel = cx + dx;
          if (xrel < 0 || xrel >= meta.cells.x)
            continue;

          // Get relative cell from relative x and y coordinates 
          int rel_cell = xrel + yrel * meta.cells.x;
          size_t rel_start = start[rel_cell];
          size_t rel_end   = end[rel_cell];

          for (size_t p = rel_start; p < rel_end; p++)
          {
            int jdx = entries[p].pidx; 
            // Skip self
            if (jdx == idx)
              continue;   

            // relative position 
            float2 xj = particles->x[jdx];
            float2 d = make_float2(x.x - xj.x, x.y - xj.y);
            // Check within smoothing radius
            if (d.x * d.x + d.y * d.y > particles->h * particles->h)
              continue; 

            // Pointer is one more than current neighbor count 
            int ptr = atomicAdd(&neighbor_counts[idx], 1);
            // Check if less than max neighbor count and set 
            if (ptr < MN)
              neighbor_list[idx * MN + ptr] = jdx;
          }
        }
      }
    }
    __syncthreads();
  }
}


// Builds a neighbor list
__host__ void neighbor_host(ParticleMatrix* particles, Spatial table, const Metadata meta, int* neighbor_counts, int* neighbor_list)
{
  const size_t N = meta.N; 
  hipMemsetAsync(neighbor_counts, 0, N * sizeof(int), 0);    // Set to 0 agnostic to whether it's already been done
  
  // Ensure shared memory spreads well for each block 
  constexpr size_t RED_BLOCKSIZE = BLOCKSIZE / 2;
  const size_t GRIDSIZE = meta.cells.x * meta.cells.y; 
  const size_t shared_memory = RED_BLOCKSIZE * sizeof(size_t);
  neighbor_search<<<GRIDSIZE, RED_BLOCKSIZE, shared_memory>>>(
    particles, 
    meta, 
    table.entries,
    table.start,
    table.end,
    neighbor_counts,
    neighbor_list
  );
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
}

// Smoothing Kernel functions 
namespace kern 
{

__constant__ float poly6_constant = 4.0 / 3.1415926;

// poly6 2d density kernel
// Since r isn't explicitly required, opted to only use the squared value of r 
__device__ float poly6(float sqr, float sqh)
{
  float sqd = sqh - sqr;
  // Outside influence - shouldn't occur
  if (sqd <= 0.0)
    return 0.0;

  float C = poly6_constant / powf(sqh, 4);
  return C * sqd * sqd * sqd;
}


// Gradiant of poly6 smoothing kernel 
__device__ float2 poly6grad(float2 r, float sqh)
{
  // Compute squared distance 
  float sqr = r.x * r.x + r.y * r.y;
  float sqd = sqh - sqr;  

  if (sqd <= 0.0)
    return make_float2(0.0, 0.0);

  float C = poly6_constant / powf(sqh, 4);
  float coeff = -6.0 * C * sqd * sqd; 
  // Return distance vector scaled 
  return make_float2(r.x * coeff, r.y * coeff);
}


// Laplacian of poly6 smoothing kernel
__device__ float poly6laplacian(float sqr, float sqh)
{
  float sqd = sqh - sqr; 
  if (sqd <= 0.0)
    return 0.0;

  float C = poly6_constant / powf(sqh, 4);
  return -12.0 * C * sqd * (sqh - 3.0 * sqr);
}

}


// Computes all accumulated density for each particle 
__global__ void compute_densities(ParticleMatrix* particles, const int* neighbor_counts, const int* neighbor_list)
{
  const size_t idx = threadIdx.x + blockDim.x * blockIdx.x; 
  if (idx >= particles->cols)
    return;

  float sum = 0.0; 
  float sqh = particles->h * particles->h;
  // Iterate over num of neighbors 
  for (int num = 0; num < neighbor_counts[idx]; num++)
  {
    // Idx of relative particle from list  
    int jdx   = neighbor_list[idx * MN + num];

    // Get distance 
    float2 d  = make_float2(
      particles->x[idx].x - particles->x[jdx].x,
      particles->x[idx].y - particles->x[jdx].y
    );

    // Compute density from kernel 
    float sqr = d.x * d.x + d.y * d.y; 
    sum += particles->mass * kern::poly6(sqr, sqh);
  }

  // Set accumulated density 
  particles->density[idx] = sum;
}


// Density precomputed - Compute all forces  
__global__ void compute_forces(ParticleMatrix* particles, const int* neighbor_counts, const int* neighbor_list)
{
  const size_t idx = threadIdx.x + blockDim.x * blockIdx.x; 
  if (idx >= particles->cols)
    return; 

  // Pre-compute ? 
  const float kb = rho0 * c0 * c0; 
  const float kd = 2 * sqrt(kb * particles->mass);

  // Get local values for particle 
  const float2 x      = particles->x[idx];
  const float xrho    = particles->density[idx]; 
  const float xpres   = (rho0 * c0 * c0) * (xrho - rho0);
  const float2 v      = particles->v[idx];

  particles->fsys[idx].y += particles->mass * -9.81; 

  // Check for x boundary 
  if (x.x < particles->h)
    particles->fsys[idx].x += kb * (particles->h - x.x) - (kd * v.x);
  else if (x.x > L - particles->h)
    particles->fsys[idx].x += -kb * (particles->h -x.x) - (kd * v.x);
  
  // Likewise check y boundary 
  if (x.y < particles->h)
    particles->fsys[idx].y += kb * (particles->h - x.y) - (kd * v.y);
  else if (x.y > L - particles->h)
    particles->fsys[idx].y += -kb * (particles->h -x.y) - (kd * v.y);

  for (int num = 0; num < neighbor_counts[idx]; num++)
{
    // Get relative values for neighbor 
    size_t jdx  = neighbor_list[idx * MN + num];
    float2 xrel = particles->x[jdx];
    float2 dj   = make_float2(x.x - xrel.x, x.y - xrel.y);

    // Compute pressure from density for relative particle
    float jrho = particles->density[jdx];
    float jpres = (rho0 * c0 * c0) * (jrho - rho0);

    // Compute the pressure force 
    float a = -particles->mass * ((xpres / (xrho * xrho)) + (jpres / (jrho * jrho)));
    float2 gW = kern::poly6grad(dj, particles->h * particles->h);
    float2 av = make_float2(gW.x * a, gW.y * a);
    particles->fpres[idx] = add_float2(particles->fpres[idx], av);

    // Compute the viscosity force 
    float laplacian = kern::poly6laplacian(dj.x * dj.x + dj.y * dj.y, particles->h * particles->h);
    float b = visc * particles->mass / jrho * laplacian;
    float2 relv = subtract_float2(particles->v[jdx], v);
    particles->fvisc[idx] = add_float2(particles->fvisc[idx], make_float2(relv.x * b, relv.y * b));
  }
}


// Computes first half of integration with half a timestep 
__global__ void first_integrate(ParticleMatrix* particles, float half_dt)
{
  const size_t idx = threadIdx.x + blockDim.x * blockIdx.x; 
  if (idx >= particles->cols)
    return;

  // sum total forces
  float2 ftotal = add_float2(
    particles->fsys[idx], 
    add_float2(particles->fpres[idx], particles->fvisc[idx])
  );

  // Compute acceleration from force and integrate for a half step
  float2 a = make_float2(ftotal.x / particles->density[idx], ftotal.y / particles->density[idx]);
  particles->v[idx].x += a.x * half_dt;
  particles->v[idx].y += a.y * half_dt; 
}


// Computes second half after forces have been calculated 
__global__ void second_integrate(ParticleMatrix* particles, float half_dt)
{
  const size_t idx = threadIdx.x + blockDim.x * blockIdx.x; 
  if (idx >= particles->cols)
    return;

  particles->x[idx].x += particles->v[idx].x * half_dt;
  particles->x[idx].y += particles->v[idx].y * half_dt;
}


__global__ void reset_accumulators(ParticleMatrix* particles)
{
  const size_t idx = threadIdx.x + blockDim.x * blockIdx.x; 
  if (idx >= particles->cols)
    return;

  if (particles->x[idx].x > L || particles->x[idx].x < 0 || particles->x[idx].y > L || particles->x[idx].y < 0)
    printf("Particle out of bounds (%f,%f): %lu\n", particles->x[idx].x, particles->x[idx].y, idx);

  // Zero out accumulated values 
  particles->density[idx] = 0.0;
  particles->fsys[idx] = particles->fpres[idx] = particles->fvisc[idx] = zero_vector; 
}


// Compute the forces given the filled neighbor counts and list
// Don't need table. 
__host__ void handle_forces(ParticleMatrix* particles, const Metadata meta, int* neighbor_counts, int* neighbor_list)
{
  // TODO: Compute dynamic timestep 
  const float dt = 1e-4 / 2.0; // dt(particles) -> float  
  const size_t GRIDSIZE = (meta.N + BLOCKSIZE - 1) / BLOCKSIZE;

  // KERNEL CALLS - All use the same launch parameters  
  reset_accumulators<<<GRIDSIZE, BLOCKSIZE>>>(particles);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  compute_densities<<<GRIDSIZE, BLOCKSIZE>>>(particles, neighbor_counts, neighbor_list);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  compute_forces<<<GRIDSIZE, BLOCKSIZE>>>(particles, neighbor_counts, neighbor_list);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  first_integrate<<<GRIDSIZE, BLOCKSIZE>>>(particles, dt); 
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  
  second_integrate<<<GRIDSIZE, BLOCKSIZE>>>(particles, dt);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  compute_densities<<<GRIDSIZE, BLOCKSIZE>>>(particles, neighbor_counts, neighbor_list);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  compute_forces<<<GRIDSIZE, BLOCKSIZE>>>(particles, neighbor_counts, neighbor_list);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  first_integrate<<<GRIDSIZE, BLOCKSIZE>>>(particles, dt); 
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
}


namespace bufr {

// Struct to hold buffer 
struct Buffer 
{
  GLuint pos_vbo{0};
  GLuint rho_vbo{0};
  hipGraphicsResource* pos_res{nullptr};
  hipGraphicsResource* rho_res{nullptr};
};


// Takes number of particles and sets up buffers
void initialize_cuda_buffers(Buffer* buffers, size_t N)
{
  // Setup buffer for position and allocate resources from cuda 
  glGenBuffers(1, &buffers->pos_vbo);
  glBindBuffer(GL_ARRAY_BUFFER, buffers->pos_vbo);
  glBufferData(GL_ARRAY_BUFFER, N * sizeof(float2), nullptr, GL_DYNAMIC_DRAW); 
  glBindBuffer(GL_ARRAY_BUFFER, 0);
  hipGraphicsGLRegisterBuffer(&buffers->pos_res, buffers->pos_vbo, hipGraphicsRegisterFlagsNone);

  // And density 
  glGenBuffers(1, &buffers->rho_vbo);
  glBindBuffer(GL_ARRAY_BUFFER, buffers->rho_vbo);
  glBufferData(GL_ARRAY_BUFFER, N * sizeof(float), nullptr, GL_DYNAMIC_DRAW); 
  glBindBuffer(GL_ARRAY_BUFFER, 0);
  hipGraphicsGLRegisterBuffer(&buffers->rho_res, buffers->rho_vbo, hipGraphicsRegisterFlagsNone);
}


void update_buffers(Buffer* buffers, ParticleMatrix host_particles)
{
  size_t size; 
  float2* d_pos;
  float* d_rho;

  hipGraphicsMapResources(1, &buffers->pos_res, 0);
  hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&d_pos), &size, buffers->pos_res);
  hipMemcpy(d_pos, host_particles.x, host_particles.cols * sizeof(float2), hipMemcpyDeviceToDevice);
  hipGraphicsUnmapResources(1, &buffers->pos_res, 0);

  hipGraphicsMapResources(1, &buffers->rho_res, 0);
  hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&d_rho), &size, buffers->rho_res);
  hipMemcpy(d_rho, host_particles.density, host_particles.cols * sizeof(float), hipMemcpyDeviceToDevice);
  hipGraphicsUnmapResources(1, &buffers->rho_res, 0);
}


// Free allocated resourcess 
void unregister_buffers(Buffer* buffers)
{
  // Check for nullptr and unregister 
  if (buffers->pos_res != nullptr)
  {
    hipGraphicsUnregisterResource(buffers->pos_res);
    glDeleteBuffers(1, &buffers->pos_vbo);
    buffers->pos_res = nullptr;
  }

  // Again for density 
  if (buffers->rho_res != nullptr)
  {
    hipGraphicsUnregisterResource(buffers->rho_res);
    glDeleteBuffers(1, &buffers->rho_vbo);
    buffers->rho_res = nullptr;
  }
}

}


namespace shdr {


// Read all data from shader file into return value 
// Credit ChatGPT o4-mini-high
std::string load_shader(const std::string& path)
{
  std::ifstream shader_file(path, std::ios::in | std::ios::binary);
  if (!shader_file)
    throw std::runtime_error("File doesn't exist");

  std::string shader; 
  shader_file.seekg(0, std::ios::end);
  shader.resize(shader_file.tellg());
  shader_file.seekg(0, std::ios::beg);
  shader_file.read(&shader[0], shader.size());
  shader_file.close();

  return shader;
}


// Compile the shader
GLuint compile_shader(GLenum type, const std::string& src)
{
  char error[512];
  GLint status;
  GLuint shader = glCreateShader(type);
  const char* cstr = src.c_str();

  glShaderSource(shader, 1, &cstr, nullptr);
  glCompileShader(shader);
  glGetShaderiv(shader, GL_COMPILE_STATUS, &status);
  if (!status)
  {
    glGetShaderInfoLog(shader, 512, nullptr, error); 
    throw std::runtime_error(error);
  }

  return shader;
}


// Creates the program from file paths to fragment and vertex shaders 
GLuint create_program(const std::string& fragment_path, const std::string& vertex_path)
{
  // Get shaders from source and compile 
  std::string vertex_src   = load_shader(vertex_path);
  std::string fragment_src = load_shader(fragment_path);
  GLuint vertex_shader   = compile_shader(GL_VERTEX_SHADER, vertex_src);
  GLuint fragment_shader = compile_shader(GL_FRAGMENT_SHADER, fragment_src);
  GLuint program = glCreateProgram();

  GLint status;
  char error[512];

  // Attach shaders to program 
  glAttachShader(program, vertex_shader);
  glAttachShader(program, fragment_shader);
  glLinkProgram(program);

  glGetProgramiv(program, GL_LINK_STATUS, &status);
  if (!status)
  {
    glGetProgramInfoLog(program, 512, nullptr, error);
    glDeleteProgram(program);
    throw std::runtime_error(error);
  }

  glDeleteShader(vertex_shader);
  glDeleteShader(fragment_shader);

  return program;
}

}

// Main 
int main(void)
{
  // Create opengl context 
  InitWindow(800, 800, "SPH");
  glewInit();
  glEnable(GL_PROGRAM_POINT_SIZE);

  GLuint program = shdr::create_program("shaders/fragment.frag", "shaders/vertex.vert");
  GLint projection_location = glGetUniformLocation(program, "uProj");

  GLuint point_vao;
  glGenVertexArrays(1, &point_vao);
  glBindVertexArray(point_vao);

  // System constants
  const size_t N = 100; 
  const size_t M = std::ceil(std::sqrt(N));
  const float particle_spacing = L_host / M; 
  const float h = 2.0 * particle_spacing; 

  // Set metadata
  Metadata meta = (Metadata)
  {
    .cells = make_uint2(std::ceil(L_host / h), std::ceil(L_host / h)),
    .N     = N
  };

  // Allocate memory to particle device ptr
  ParticleMatrix h_p, *particles;
  hipMalloc(&h_p.density, N * sizeof(float));
  hipMalloc(&h_p.x, N * sizeof(float2)); 
  hipMalloc(&h_p.v, N * sizeof(float2));
  hipMalloc(&h_p.fpres, N * sizeof(float2)); 
  hipMalloc(&h_p.fvisc, N * sizeof(float2));
  hipMalloc(&h_p.fsys, N * sizeof(float2));
  hipMalloc(&particles, sizeof(ParticleMatrix));

  // Set constant values 
  h_p.cols = N;
  h_p.mass = rho0_host * particle_spacing * particle_spacing; 
  h_p.h    = h;
  
  // Particle position initialization
  // Uniform distribution over range [h, L - h]
  float exact_dx = (L_host - 2 * h) / N; 
  int cols = static_cast<int>(std::floor(L_host - 2 * h / exact_dx));
  int rows = static_cast<int>(std::ceil(static_cast<float>(N) / cols)); 

  std::vector<float2> host_positions(N);
  std::vector<float2> host_velocities(N);

  // Initialize position in square lattice 
  int pid = 0; 
  for (int i = 0; i < rows && pid < N; i++)
  {
    for (int j = 0; j < cols && pid < N; j++)
    {
      float x = h + (i + 0.5) * exact_dx; 
      float y = h + (j + 0.5) * exact_dx; 
      host_positions[pid] = make_float2(x, y);
      host_velocities[pid] = zero_vec_host;
      pid++;
    }
  }

  // Copy initialized vectors to device  
  hipMemcpy(h_p.x, host_positions.data(), N * sizeof(float2), hipMemcpyHostToDevice);
  hipMemcpy(h_p.v, host_velocities.data(), N * sizeof(float2), hipMemcpyHostToDevice);

  // Copy all to device pointer 
  hipMemcpy(particles, &h_p, sizeof(ParticleMatrix), hipMemcpyHostToDevice);

  // Table is already a host pointer so no memcpy required to a device pointer
  Spatial table;
  table.cells = meta.cells; 
  hipMalloc(&table.entries, N * sizeof(Spatial::value));
  hipMalloc(&table.start, meta.cells.x * meta.cells.y * sizeof(size_t));
  hipMalloc(&table.end, meta.cells.x * meta.cells.y * sizeof(size_t));

  // Allocate memory for lists
  int *neighbor_list, *neighbor_counts; 
  hipMalloc(&neighbor_counts, N * sizeof(int));
  hipMalloc(&neighbor_list, N * MN_host * sizeof(int));

  // setting up matrix 
  float projection_matrix[16] = {
    2.0/L_host,      0,    0,    0,
        0, -2.0/L_host,    0,    0,
        0,      0,   -1,    0,
     -1.0,    1.0,    0,    1
  };

  // Set up buffers 
  bufr::Buffer buffer;
  bufr::initialize_cuda_buffers(&buffer, N);

  // Simulation loop
  SetTargetFPS(144);
  while (!WindowShouldClose())
  {

    // One timestep of simulation  
    generate_spatial_table(particles, table, meta);
    neighbor_host(particles, table, meta, neighbor_counts, neighbor_list);
    handle_forces(particles, meta, neighbor_counts, neighbor_list);

    bufr::update_buffers(&buffer, h_p);

    // Handle drawing from buffer 

    BeginDrawing();
      ClearBackground(BLACK);

      DrawRectangleLines(100, 100, 600, 600, WHITE);
      glViewport(100, 100, 600, 600);

      glBindVertexArray(point_vao);
      glUseProgram(program);
      glUniformMatrix4fv(projection_location, 1, GL_FALSE, projection_matrix);

      glEnableVertexAttribArray(0);
      glBindBuffer(GL_ARRAY_BUFFER, buffer.pos_vbo);
      glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, 0, (void*)0);
      glDrawArrays(GL_POINTS, 0, N);

      glDisableVertexAttribArray(0);
      glBindBuffer(GL_ARRAY_BUFFER, 0);
      glBindVertexArray(0);
      glUseProgram(0);
      glViewport(0, 0, 800, 800);

    EndDrawing();
  }

  // Free resources 

  glDeleteProgram(program);
  CloseWindow();
  bufr::unregister_buffers(&buffer);

  hipFree(neighbor_counts);
  hipFree(neighbor_list); 
  hipFree(table.entries);
  hipFree(table.start);
  hipFree(table.end);

  // Copy particles back to host to free resources 
  hipMemcpy(&h_p, particles, sizeof(ParticleMatrix), hipMemcpyDeviceToHost);

  hipFree(h_p.x);
  hipFree(h_p.density);
  hipFree(h_p.fsys);
  hipFree(h_p.fpres);
  hipFree(h_p.fvisc);
  hipFree(particles);

  return 0; 
}
